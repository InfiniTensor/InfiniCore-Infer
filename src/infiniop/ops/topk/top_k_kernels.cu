#include "hip/hip_runtime.h"
/*
 * Adapted from https://github.com/NVIDIA/TensorRT-LLM/blob/v0.7.1/cpp/tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.cu
 * Copyright (c) 2024, The vLLM team.
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cub/util_type.cuh>
#include <hipcub/hipcub.hpp>
#include <cfloat>

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define WARP_SIZE 32

#define VLLM_SHFL_XOR_SYNC_WIDTH(VAL, MASK, WIDTH) __shfl_xor_sync(0xFFFFFFFF, VAL, MASK, WIDTH)


 // Aligned array type
 template <
	 typename T,
	 int N,
	 int Alignment = sizeof(T) * N
 >
 class alignas(Alignment) AlignedArray {
	 float data[N];
 };

template <int TPB>
__launch_bounds__(TPB) __global__
void moeSoftmax(const float* input, const bool* finished, float* output, const int num_cols)
{
    using BlockReduce = hipcub::BlockReduce<float, TPB>;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    __shared__ float normalizing_factor;
    __shared__ float float_max;

    const int thread_row_offset = blockIdx.x * num_cols;

    hipcub::Sum sum;
    float threadData(-FLT_MAX);

    if ((finished != nullptr) && finished[blockIdx.x])
    {
        return;
    }

    for (int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx = thread_row_offset + ii;
        threadData = max(static_cast<float>(input[idx]), threadData);
    }

    const float maxElem = BlockReduce(tmpStorage).Reduce(threadData, hipcub::Max());
    if (threadIdx.x == 0)
    {
        float_max = maxElem;
    }
    __syncthreads();

    threadData = 0;

    for (int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx = thread_row_offset + ii;
        threadData += expf((static_cast<float>(input[idx]) - float_max));
    }

    const auto Z = BlockReduce(tmpStorage).Reduce(threadData, sum);

    if (threadIdx.x == 0)
    {
        normalizing_factor = 1.f / Z;
    }
    __syncthreads();

    for (int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx = thread_row_offset + ii;
        const float val = expf((static_cast<float>(input[idx]) - float_max)) * normalizing_factor;
        output[idx] = val;
    }
}

template <int TPB, typename IndType>
__launch_bounds__(TPB) __global__ void moeTopK(
    const float* inputs_after_softmax,
    const bool* finished,
    float* output,
    IndType* indices,
    int* source_rows,
    const int num_experts,
    const int k,
    const int start_expert,
    const int end_expert)
{

    using cub_kvp = hipcub::KeyValuePair<int, float>;
    using BlockReduce = hipcub::BlockReduce<cub_kvp, TPB>;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    cub_kvp thread_kvp;
    hipcub::ArgMax arg_max;

    const int num_rows = gridDim.x;
    const int block_row = blockIdx.x;

    const bool row_is_active = finished ? !finished[block_row] : true;
    const int thread_read_offset = blockIdx.x * num_experts;
    for (int k_idx = 0; k_idx < k; ++k_idx)
    {
        thread_kvp.key = 0;
        thread_kvp.value = -1.f; // This is OK because inputs are probabilities

        cub_kvp inp_kvp;
        for (int expert = threadIdx.x; expert < num_experts; expert += TPB)
        {
            const int idx = thread_read_offset + expert;
            inp_kvp.key = expert;
            inp_kvp.value = inputs_after_softmax[idx];

            for (int prior_k = 0; prior_k < k_idx; ++prior_k)
            {
                const int prior_winning_expert = indices[k * block_row + prior_k];

                if (prior_winning_expert == expert)
                {
                    inp_kvp = thread_kvp;
                }
            }

            thread_kvp = arg_max(inp_kvp, thread_kvp);
        }

        const cub_kvp result_kvp = BlockReduce(tmpStorage).Reduce(thread_kvp, arg_max);
        if (threadIdx.x == 0)
        {
            const int expert = result_kvp.key;
            const bool node_uses_expert = expert >= start_expert && expert < end_expert;
            const bool should_process_row = row_is_active && node_uses_expert;

            const int idx = k * block_row + k_idx;
            output[idx] = result_kvp.value;
            indices[idx] = should_process_row ? (expert - start_expert) : num_experts;
            if (source_rows)
                source_rows[idx] = k_idx * num_rows + block_row;
        }
        __syncthreads();
    }
}

template <int VPT, int NUM_EXPERTS, int WARPS_PER_CTA, int BYTES_PER_LDG, typename IndType>
__launch_bounds__(WARPS_PER_CTA* WARP_SIZE) __global__
void topkGatingSoftmax(const float* input, const bool* finished, float* output, const int num_rows, IndType* indices,
    int* source_rows, const int k, const int start_expert, const int end_expert)
{
    static_assert(VPT == (VPT & -VPT), "VPT must be power of 2");
    static_assert(NUM_EXPERTS == (NUM_EXPERTS & -NUM_EXPERTS), "NUM_EXPERTS must be power of 2");
    static_assert(BYTES_PER_LDG == (BYTES_PER_LDG & -BYTES_PER_LDG), "BYTES_PER_LDG must be power of 2");
    static_assert(BYTES_PER_LDG <= 16, "BYTES_PER_LDG must be leq 16");

    static constexpr int ELTS_PER_LDG = BYTES_PER_LDG / sizeof(float);
    static constexpr int ELTS_PER_ROW = NUM_EXPERTS;
    static constexpr int THREADS_PER_ROW = ELTS_PER_ROW / VPT;
    static constexpr int LDG_PER_THREAD = VPT / ELTS_PER_LDG;

    static_assert(VPT % ELTS_PER_LDG == 0, "The elements per thread must be a multiple of the elements per ldg");
    static_assert(WARP_SIZE % THREADS_PER_ROW == 0, "The threads per row must cleanly divide the threads per warp");
    static_assert(THREADS_PER_ROW == (THREADS_PER_ROW & -THREADS_PER_ROW), "THREADS_PER_ROW must be power of 2");
    static_assert(THREADS_PER_ROW <= WARP_SIZE, "THREADS_PER_ROW can be at most warp size");

    static constexpr int ELTS_PER_WARP = WARP_SIZE * VPT;
    static constexpr int ROWS_PER_WARP = ELTS_PER_WARP / ELTS_PER_ROW;
    static constexpr int ROWS_PER_CTA = WARPS_PER_CTA * ROWS_PER_WARP;

    static_assert(ELTS_PER_WARP % ELTS_PER_ROW == 0, "The elts per row must cleanly divide the total elt per warp");

    const int cta_base_row = blockIdx.x * ROWS_PER_CTA;
    const int warp_base_row = cta_base_row + threadIdx.y * ROWS_PER_WARP;
    const int thread_row_in_warp = threadIdx.x / THREADS_PER_ROW;
    const int thread_row = warp_base_row + thread_row_in_warp;

    if (thread_row >= num_rows)
    {
        return;
    }
    const bool row_is_active = finished ? !finished[thread_row] : true;

    const float* thread_row_ptr = input + thread_row * ELTS_PER_ROW;

    const int thread_group_idx = threadIdx.x % THREADS_PER_ROW;
    const int first_elt_read_by_thread = thread_group_idx * ELTS_PER_LDG;
    const float* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;

    using AccessType = AlignedArray<float, ELTS_PER_LDG>;

    float row_chunk[VPT];
    AccessType* row_chunk_vec_ptr = reinterpret_cast<AccessType*>(&row_chunk);
    const AccessType* vec_thread_read_ptr = reinterpret_cast<const AccessType*>(thread_read_ptr);
#pragma unroll
    for (int ii = 0; ii < LDG_PER_THREAD; ++ii)
    {
        row_chunk_vec_ptr[ii] = vec_thread_read_ptr[ii * THREADS_PER_ROW];
    }

    float thread_max = row_chunk[0];
#pragma unroll
    for (int ii = 1; ii < VPT; ++ii)
    {
        thread_max = max(thread_max, row_chunk[ii]);
    }

#pragma unroll
    for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2)
    {
        thread_max = max(thread_max, VLLM_SHFL_XOR_SYNC_WIDTH(thread_max, mask, THREADS_PER_ROW));
    }

    float row_sum = 0;
#pragma unroll
    for (int ii = 0; ii < VPT; ++ii)
    {
        row_chunk[ii] = expf(row_chunk[ii] - thread_max);
        row_sum += row_chunk[ii];
    }

#pragma unroll
    for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2)
    {
        row_sum += VLLM_SHFL_XOR_SYNC_WIDTH(row_sum, mask, THREADS_PER_ROW);
    }

    const float reciprocal_row_sum = 1.f / row_sum;

#pragma unroll
    for (int ii = 0; ii < VPT; ++ii)
    {
        row_chunk[ii] = row_chunk[ii] * reciprocal_row_sum;
    }

    int start_col = first_elt_read_by_thread;
    static constexpr int COLS_PER_GROUP_LDG = ELTS_PER_LDG * THREADS_PER_ROW;

    for (int k_idx = 0; k_idx < k; ++k_idx)
    {
        float max_val = row_chunk[0];
        int expert = start_col;
#pragma unroll
        for (int ldg = 0, col = start_col; ldg < LDG_PER_THREAD; ++ldg, col += COLS_PER_GROUP_LDG)
        {
#pragma unroll
            for (int ii = 0; ii < ELTS_PER_LDG; ++ii)
            {
                float val = row_chunk[ldg * ELTS_PER_LDG + ii];

                if (val > max_val)
                {
                    max_val = val;
                    expert = col + ii;
                }
            }
        }

#pragma unroll
        for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2)
        {
            float other_max = VLLM_SHFL_XOR_SYNC_WIDTH(max_val, mask, THREADS_PER_ROW);
            int other_expert = VLLM_SHFL_XOR_SYNC_WIDTH(expert, mask, THREADS_PER_ROW);

            if (other_max > max_val || (other_max == max_val && other_expert < expert))
            {
                max_val = other_max;
                expert = other_expert;
            }
        }

        if (thread_group_idx == 0)
        {
            const bool node_uses_expert = expert >= start_expert && expert < end_expert;
            const bool should_process_row = row_is_active && node_uses_expert;

            const int idx = k * thread_row + k_idx;
            output[idx] = max_val;
            indices[idx] = should_process_row ? (expert - start_expert) : NUM_EXPERTS;
            if (source_rows)
                source_rows[idx] = k_idx * num_rows + thread_row;
        }

        if (k_idx + 1 < k)
        {
            const int ldg_group_for_expert = expert / COLS_PER_GROUP_LDG;
            const int thread_to_clear_in_group = (expert / ELTS_PER_LDG) % THREADS_PER_ROW;

            if (thread_group_idx == thread_to_clear_in_group)
            {
                const int offset_for_expert = expert % ELTS_PER_LDG;
                row_chunk[ldg_group_for_expert * ELTS_PER_LDG + offset_for_expert] = -10000.f;
            }
        }
    }
}

namespace detail
{
template <int EXPERTS, int BYTES_PER_LDG>
struct TopkConstants
{
    static constexpr int ELTS_PER_LDG = BYTES_PER_LDG / sizeof(float);
    static_assert(EXPERTS / (ELTS_PER_LDG * WARP_SIZE) == 0 || EXPERTS % (ELTS_PER_LDG * WARP_SIZE) == 0, "");
    static constexpr int VECs_PER_THREAD = MAX(1, EXPERTS / (ELTS_PER_LDG * WARP_SIZE));
    static constexpr int VPT = VECs_PER_THREAD * ELTS_PER_LDG;
    static constexpr int THREADS_PER_ROW = EXPERTS / VPT;
    static constexpr int ROWS_PER_WARP = WARP_SIZE / THREADS_PER_ROW;
};
}

template <int EXPERTS, int WARPS_PER_TB, typename IndType>
void topkGatingSoftmaxLauncherHelper(const float* input, const bool* finished, float* output, IndType* indices,
    int* source_row, const int num_rows, const int k, const int start_expert, const int end_expert, hipStream_t stream)
{
    static constexpr std::size_t MAX_BYTES_PER_LDG = 16;

    static constexpr int BYTES_PER_LDG = MIN(MAX_BYTES_PER_LDG, sizeof(float) * EXPERTS);
    using Constants = detail::TopkConstants<EXPERTS, BYTES_PER_LDG>;
    static constexpr int VPT = Constants::VPT;
    static constexpr int ROWS_PER_WARP = Constants::ROWS_PER_WARP;
    const int num_warps = (num_rows + ROWS_PER_WARP - 1) / ROWS_PER_WARP;
    const int num_blocks = (num_warps + WARPS_PER_TB - 1) / WARPS_PER_TB;

    dim3 block_dim(WARP_SIZE, WARPS_PER_TB);
    topkGatingSoftmax<VPT, EXPERTS, WARPS_PER_TB, BYTES_PER_LDG><<<num_blocks, block_dim, 0, stream>>>(
        input, finished, output, num_rows, indices, source_row, k, start_expert, end_expert);
}

#define LAUNCH_SOFTMAX(NUM_EXPERTS, WARPS_PER_TB)                       \
    topkGatingSoftmaxLauncherHelper<NUM_EXPERTS, WARPS_PER_TB>(         \
        (const float*)gating_output, nullptr, (float*)topk_weights, (IndType*)topk_indices,            \
        (int*)token_expert_indices, num_tokens, topk, 0, num_experts,         \
        stream);

template <typename IndType>
void topkGatingSoftmaxKernelLauncher(
    const void* gating_output,
    void* topk_weights,
    void* topk_indices,
    void* token_expert_indices,
    void* softmax_workspace,
    const int num_tokens,
    const int num_experts,
    const int topk,
    hipStream_t stream) {
    static constexpr int WARPS_PER_TB = 4;
    switch (num_experts) {
        case 1:
            LAUNCH_SOFTMAX(1, WARPS_PER_TB);
            break;
        case 2:
            LAUNCH_SOFTMAX(2, WARPS_PER_TB);
            break;
        case 4:
            LAUNCH_SOFTMAX(4, WARPS_PER_TB);
            break;
        case 8:
            LAUNCH_SOFTMAX(8, WARPS_PER_TB);
            break;
        case 16:
            LAUNCH_SOFTMAX(16, WARPS_PER_TB);
            break;
        case 32:
            LAUNCH_SOFTMAX(32, WARPS_PER_TB);
            break;
        case 64:
            LAUNCH_SOFTMAX(64, WARPS_PER_TB);
            break;
        case 128:
            LAUNCH_SOFTMAX(128, WARPS_PER_TB);
            break;
        case 256:
            LAUNCH_SOFTMAX(256, WARPS_PER_TB);
            break;
        default: {
            static constexpr int TPB = 256;
            moeSoftmax<TPB><<<num_tokens, TPB, 0, stream>>>(
                (const float*)gating_output, nullptr, (float*)softmax_workspace, num_experts);
            moeTopK<TPB, IndType><<<num_tokens, TPB, 0, stream>>>(
                (const float*)softmax_workspace, nullptr, (float*)topk_weights, (IndType*)topk_indices, (int*)token_expert_indices,
                num_experts, topk, 0, num_experts);
        }
    }
} 