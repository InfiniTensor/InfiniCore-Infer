#include "hip/hip_runtime.h"
#include "../../../devices/nvidia/nvidia_handle.cuh"
#include "moe_dispatch.cuh"
#include "../info.h"
#include <cub/device/device_scan.cuh>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h> // CHANGE: 添加 bfloat16 头文件
#include <hip/hip_runtime.h>

namespace op::moe_dispatch::nvidia {

struct Descriptor::Opaque {
    std::shared_ptr<device::nvidia::Handle::Internal> internal;
};

Descriptor::~Descriptor() {
    delete _opaque;
}

// NOTE: create 函数无需更改，它已经正确地检查了所有需要的数据类型
infiniStatus_t Descriptor::create(
    infiniopHandle_t handle_, Descriptor **desc_ptr, int num_experts,
    infiniopTensorDescriptor_t input_desc,
    infiniopTensorDescriptor_t indices_desc,
    infiniopTensorDescriptor_t permuted_output_desc,
    infiniopTensorDescriptor_t aux_info_desc) {

    auto handle = reinterpret_cast<device::nvidia::Handle *>(handle_);
    auto dtype = input_desc->dtype();

    CHECK_DTYPE(dtype, INFINI_DTYPE_F16, INFINI_DTYPE_F32, INFINI_DTYPE_BF16);

    // NEW: Add hardware check for BF16 during descriptor creation
    if (dtype == INFINI_DTYPE_BF16) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, handle_->device_id);
        if (props.major < 8) {
            fprintf(stderr,
                    "Error: BF16 data type is not supported on this GPU "
                    "architecture (Compute Capability %d.%d). It requires "
                    "Compute Capability 8.0 (Ampere) or higher.\n",
                    props.major, props.minor);
            return INFINI_STATUS_DEVICE_TYPE_NOT_SUPPORTED;
        }
    }

    auto result = MoEDispatchInfo::create(num_experts,input_desc, indices_desc,
                                          permuted_output_desc, aux_info_desc
                                          );
    CHECK_RESULT(result);

    *desc_ptr = new Descriptor(result.take(), new Opaque{handle->internal()},
                               handle->device, handle_->device_id);
    return INFINI_STATUS_SUCCESS;
}

// NOTE: count_experts_kernel 无需更改，因为它只处理整数索引，与数据类型无关
__global__ void count_experts_kernel(const int *indices, int *expert_counts,
                                     int num_tokens, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_tokens * k) {
        int expert_idx = indices[i];
        if (expert_idx >= 0) { // Assuming invalid experts are marked as < 0
            atomicAdd(&expert_counts[expert_idx], 1);
        }
    }
}

// NOTE: dispatch_kernel 无需更改，因为它是一个模板核函数，本身就是类型通用的
template <typename T, typename IndT>
__global__ void dispatch_kernel(const T *input, const IndT *indices,
                                T *permuted_output, int *expert_offsets,
                                int *aux_info, int num_tokens, int k,
                                int hidden_dim) {
    int token_idx = blockIdx.x;
    int data_idx = threadIdx.x;

    if (token_idx >= num_tokens || data_idx >= hidden_dim)
        return;

    for (int i = 0; i < k; ++i) {
        int expert_idx = indices[token_idx * k + i];
        if (expert_idx >= 0) {
            int dispatch_pos = atomicAdd(&expert_offsets[expert_idx], 1);
            permuted_output[dispatch_pos * hidden_dim + data_idx] =
                input[token_idx * hidden_dim + data_idx];
            if (data_idx == 0) {
                aux_info[dispatch_pos * 2 + 0] = token_idx;
                aux_info[dispatch_pos * 2 + 1] = token_idx * k + i;
            }
        }
    }
}

// CHANGE: 对 calculate 函数进行了大幅修改
infiniStatus_t Descriptor::calculate(const void *input, const void *indices,
                                     void *permuted_output, void *aux_info,
                                     void *stream) const {
    // 只检查索引类型，数据类型的检查将在 switch 中处理
    if (_info.index_type != INFINI_DTYPE_I32) {
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    int *expert_counts, *expert_offsets;
    size_t count_size = _info.num_experts * sizeof(int);
    hipMalloc(&expert_counts, count_size);
    hipMalloc(&expert_offsets, count_size);
    hipMemsetAsync(expert_counts, 0, count_size, (hipStream_t)stream);

    dim3 count_grid((_info.num_tokens * _info.k + 255) / 256);
    dim3 count_block(256);
    count_experts_kernel<<<count_grid, count_block, 0, (hipStream_t)stream>>>(
        (const int *)indices, expert_counts, _info.num_tokens, _info.k);

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  expert_counts, expert_offsets,
                                  _info.num_experts);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  expert_counts, expert_offsets,
                                  _info.num_experts, (hipStream_t)stream);

    int *dispatch_offsets;
    hipMalloc(&dispatch_offsets, count_size);
    hipMemcpyAsync(dispatch_offsets, expert_offsets, count_size,
                    hipMemcpyDeviceToDevice, (hipStream_t)stream);

    dim3 dispatch_grid(_info.num_tokens);
    dim3 dispatch_block(_info.hidden_dim);
    if (_info.hidden_dim > 1024)
        dispatch_block.x = 1024;

    // NEW: 使用 switch 语句根据数据类型分发不同模板的 Kernel
    switch (_info.data_type) {
    case INFINI_DTYPE_F32:
        dispatch_kernel<float, int><<<dispatch_grid, dispatch_block, 0, (hipStream_t)stream>>>(
            (const float *)input, (const int *)indices, (float *)permuted_output,
            dispatch_offsets, (int *)aux_info, _info.num_tokens, _info.k,
            _info.hidden_dim);
        break;

    case INFINI_DTYPE_F16:
        dispatch_kernel<__half, int><<<dispatch_grid, dispatch_block, 0, (hipStream_t)stream>>>(
            (const __half *)input, (const int *)indices, (__half *)permuted_output,
            dispatch_offsets, (int *)aux_info, _info.num_tokens, _info.k,
            _info.hidden_dim);
        break;

    case INFINI_DTYPE_BF16:
        dispatch_kernel<__hip_bfloat16, int><<<dispatch_grid, dispatch_block, 0, (hipStream_t)stream>>>(
            (const __hip_bfloat16 *)input, (const int *)indices, (__hip_bfloat16 *)permuted_output,
            dispatch_offsets, (int *)aux_info, _info.num_tokens, _info.k,
            _info.hidden_dim);
        break;

    default:
        // 释放已分配的内存
        hipFree(d_temp_storage);
        hipFree(expert_counts);
        hipFree(expert_offsets);
        hipFree(dispatch_offsets);
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    hipFree(d_temp_storage);
    hipFree(expert_counts);
    hipFree(expert_offsets);
    hipFree(dispatch_offsets);
    return INFINI_STATUS_SUCCESS;
}

} // namespace op::moe_dispatch::nvidia